#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <filesystem/directory.h>
#include <filesystem/path.h>
#include <nerf-cuda/common.h>
#include <nerf-cuda/nerf_network.h>
#include <nerf-cuda/nerf_render.h>
#include <nerf-cuda/render_utils.h>
#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/gpu_matrix.h>
#include <tiny-cuda-nn/gpu_memory.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>

#include <algorithm>
#include <cmath>
#include <fstream>
#include <json/json.hpp>
#include <nerf-cuda/common_device.cuh>
#include <set>
#include <typeinfo>
#include <vector>
#include "omp.h"

using namespace Eigen;
using namespace tcnn;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN

json merge_parent_network_config(const json& child,
                                 const fs::path& child_filename) {
  if (!child.contains("parent")) {
    return child;
  }
  fs::path parent_filename =
      child_filename.parent_path() / std::string(child["parent"]);
  tlog::info() << "Loading parent network config from: "
               << parent_filename.str();
  std::ifstream f{parent_filename.str()};
  json parent = json::parse(f, nullptr, true, true);
  parent = merge_parent_network_config(parent, parent_filename);
  parent.merge_patch(child);
  return parent;
}

NerfRender::NerfRender() {
  m_network_config = {};
  m_inference_stream.resize(NGPU);
  for(uint64_t gpu=0;gpu<NGPU;gpu++){
    hipSetDevice(gpu);
    hipStreamCreate(&m_inference_stream[gpu]);

    m_density_grid.emplace_back(GPUMemory<float>(1));
    m_aabb.emplace_back(GPUMemory<float>(1));
    m_rng.emplace_back(default_rng_t{m_seed});
  }
}

NerfRender::~NerfRender() {
  delete [] deep_h;
  delete [] image_h;
  delete [] us_image;
  delete [] us_depth;
}

json NerfRender::load_network_config(const fs::path& network_config_path) {
  if (!network_config_path.empty()) {
    m_network_config_path = network_config_path;
  }

  tlog::info() << "Loading network config from: " << network_config_path;

  if (network_config_path.empty() || !network_config_path.exists()) {
    throw std::runtime_error{std::string{"Network config \""} +
                             network_config_path.str() + "\" does not exist."};
  }

  json result;
  if (equals_case_insensitive(network_config_path.extension(), "json")) {
    std::ifstream f{network_config_path.str()};
    result = json::parse(f, nullptr, true, true);
    result = merge_parent_network_config(result, network_config_path);
  } else if (equals_case_insensitive(network_config_path.extension(),
                                     "msgpack")) {
    std::ifstream f{network_config_path.str(), std::ios::in | std::ios::binary};
    result = json::from_msgpack(f);
    // we assume parent pointers are already resolved in snapshots.
  }

  return result;
}

void NerfRender::reload_network_from_file(
    const std::string& network_config_path) {
  if (!network_config_path.empty()) {
    m_network_config_path = network_config_path;
    tlog::info() << m_network_config_path.extension() ;
    if (equals_case_insensitive(m_network_config_path.extension(), "msgpack")) {
      load_snapshot(network_config_path);
      reset_network();
      for(uint64_t gpu=0;gpu<NGPU;gpu++){
        hipSetDevice(gpu);
        m_nerf_network[gpu]->deserialize(m_network_config["snapshot"]);
      }
    } else {
      throw std::runtime_error{"Input file with wrong extension!"};
    }
  }
}

void NerfRender::reset_network() {
  // Default config
  json config = m_network_config;
  json& encoding_config = config["encoding"];
  json& network_config = config["network"];
  json& dir_encoding_config = config["dir_encoding"];
  json& rgb_network_config = config["rgb_network"];
  uint32_t n_dir_dims = 3;
  uint32_t n_pos_dims = 3;
  uint32_t n_extra_dims = 0;  // Now, it's set to zero but it needs furture
                              // check! By Hangkun, 2022/06/30

  // Automatically determine certain parameters if we're dealing with the
  // (hash)grid encoding
  if (to_lower(encoding_config.value("otype", "OneBlob")).find("grid") !=
      std::string::npos) {
    encoding_config["n_pos_dims"] = n_pos_dims;  // 3 dimenison input

    const uint32_t n_features_per_level =
        encoding_config.value("n_features_per_level", 2u);
    uint32_t m_num_levels = 16u;

    if (encoding_config.contains("n_features") &&
        encoding_config["n_features"] > 0) {
      m_num_levels =
          (uint32_t)encoding_config["n_features"] / n_features_per_level;
    } else {
      m_num_levels = encoding_config.value("n_levels", 16u);
    }

    // m_level_stats.resize(m_num_levels);
    // m_first_layer_column_stats.resize(m_num_levels);

    const uint32_t log2_hashmap_size =
        encoding_config.value("log2_hashmap_size", 15);

    uint32_t m_base_grid_resolution =
        encoding_config.value("base_resolution", 0);
    if (!m_base_grid_resolution) {
      m_base_grid_resolution = 1u << ((log2_hashmap_size) / n_pos_dims);
      encoding_config["base_resolution"] = m_base_grid_resolution;
    }

    float desired_resolution = 2048.0f;  // Desired resolution of the finest
                                         // hashgrid level over the unit cube

    // Automatically determine suitable per_level_scale
    float m_per_level_scale = encoding_config.value("per_level_scale", 0.0f);
    if (m_per_level_scale <= 0.0f && m_num_levels > 1) {
      m_per_level_scale =
          std::exp(std::log(desired_resolution * (float)m_bound /
                            (float)m_base_grid_resolution) /
                   (m_num_levels - 1));
      encoding_config["per_level_scale"] = m_per_level_scale;
    }

    tlog::info() << "GridEncoding: "
                 << " Nmin=" << m_base_grid_resolution
                 << " b=" << m_per_level_scale << " F=" << n_features_per_level
                 << " T=2^" << log2_hashmap_size << " L=" << m_num_levels;
  }

  for(uint64_t gpu=0;gpu<NGPU;gpu++){
    hipSetDevice(gpu);
    // reset the random seed
    m_rng[gpu] = default_rng_t{m_seed};
    // Generate the network
    m_nerf_network.emplace_back(std::make_shared<NerfNetwork<precision_t>>(
        n_pos_dims, n_dir_dims, n_extra_dims,
        n_pos_dims,  // The offset of 1 comes from the dt member variable of
                        // NerfCoordinate. HACKY
        encoding_config, dir_encoding_config, network_config, rgb_network_config));
  }
}

void NerfRender::set_resolution(Eigen::Vector2i res)
{
  resolution = res;
  int N = res[0] * res[1] / NGPU;  // number of pixels
  for(uint64_t gpu=0;gpu<NGPU;gpu++){
    hipSetDevice(gpu);
    // initial points corresponding to pixels, in world coordination
    rays_o.emplace_back(tcnn::GPUMatrixDynamic<float>(3, N, m_inference_stream[gpu], tcnn::RM));
    // direction corresponding to pixels,in world coordination
    rays_d.emplace_back(tcnn::GPUMatrixDynamic<float>(3, N, m_inference_stream[gpu], tcnn::RM));
    // Calculate rays' intersection time (near and far) with aabb
    nears.emplace_back(tcnn::GPUMatrixDynamic<float>(1, N, m_inference_stream[gpu], tcnn::RM));
    fars.emplace_back(tcnn::GPUMatrixDynamic<float>(1, N, m_inference_stream[gpu], tcnn::RM));
    //  allocate outputs
    weight_sum.emplace_back(tcnn::GPUMatrixDynamic<float>(1, N, m_inference_stream[gpu], tcnn::RM));  // the accumlate weight of each ray
    depth.emplace_back(tcnn::GPUMatrixDynamic<float>(1, N, m_inference_stream[gpu], tcnn::RM));  // output depth img
    image.emplace_back(tcnn::GPUMatrixDynamic<float>(N, 3, m_inference_stream[gpu], tcnn::RM));  // output rgb image
    // store the alive rays number
    alive_counter.emplace_back(tcnn::GPUMatrixDynamic<int>(1, 1, m_inference_stream[gpu], tcnn::RM));
    // the alive rays' IDs in N (N >= n_alive, but we only use first n_alive)
    // 2 is used to loop old/new
    rays_alive.emplace_back(tcnn::GPUMatrixDynamic<int>(2, N, m_inference_stream[gpu], tcnn::RM));
    // the alive rays' time, we only use the first n_alive.
    // dead rays are marked by rays_t < 0
    //  2 is used to loop old/new
    rays_t.emplace_back(tcnn::GPUMatrixDynamic<float>(2, N, m_inference_stream[gpu], tcnn::RM));

    xyzs.emplace_back(tcnn::GPUMatrixDynamic<float>(3, div_round_up(N, 128) * 128, m_inference_stream[gpu], tcnn::CM));
    // all generated points' view dirs.
    dirs.emplace_back(tcnn::GPUMatrixDynamic<float>(3, div_round_up(N, 128) * 128, m_inference_stream[gpu], tcnn::CM));
    // all generated points' deltas
    //(here we record two deltas, the first is for RGB, the second for depth).
    deltas.emplace_back(tcnn::GPUMatrixDynamic<float>(2, div_round_up(N, 128) * 128, m_inference_stream[gpu], tcnn::CM));

    // volume density
    sigmas.emplace_back(tcnn::GPUMatrixDynamic<float>(1, div_round_up(N, 128) * 128, m_inference_stream[gpu], tcnn::RM));
    // emitted color
    rgbs.emplace_back(tcnn::GPUMatrixDynamic<float>(div_round_up(N, 128) * 128, 3, m_inference_stream[gpu], tcnn::RM));

    // concated input
    network_input.emplace_back(tcnn::GPUMatrixDynamic<float>(m_nerf_network[gpu]->input_width(),
                                                  div_round_up(N, 128) * 128, m_inference_stream[gpu], tcnn::RM));
    // concated output
    network_output.emplace_back(tcnn::GPUMatrixDynamic<precision_t>(
          m_nerf_network[gpu]->padded_output_width(), div_round_up(N, 128) * 128, m_inference_stream[gpu], tcnn::RM));
  }
  deep_h = new float[N*NGPU];
  image_h = new float[N*NGPU * 3];
  us_image = new unsigned char [N*NGPU*3];
  us_depth = new unsigned char [N*NGPU];
}

Image NerfRender::render_frame(struct Camera cam, Eigen::Matrix<float, 4, 4> pos) {
  // cam : parameters of cam
  // pos : camera external parameters
  // resolution : [Width, Height]

  int N = resolution[0] * resolution[1] / NGPU;  // number of pixels
  std::vector<int> step(NGPU,0);          // the current march step
  std::vector<int> i(NGPU,0);             // the flag to index old and new rays
  std::vector<bool> running(NGPU,true);
  std::vector<int> num_alive(NGPU,N);  // initialize the initial number alive as N
  threads.clear();

  // functions to generate rays_o and rays_d, it takes camera parameters and
  // resolution as input
  for(uint64_t gpu=0;gpu<NGPU;gpu++){
    threads.emplace_back(std::thread([&,gpu](){
    hipSetDevice(gpu);
    generate_rays(cam, pos, int(gpu));
    // caliucate nears and fars
    kernel_near_far_from_aabb<<<div_round_up(N, m_num_thread), m_num_thread, 0, m_inference_stream[gpu]>>>(
        rays_o[gpu].view(), rays_d[gpu].view(), m_aabb[gpu].data(), N, m_min_near, nears[gpu].view(),
        fars[gpu].view());

    // initial weight_sum, image and depth with 0
    parallel_for_gpu(m_inference_stream[gpu], weight_sum[gpu].n_elements(), [params_fp=weight_sum[gpu].data()] __device__ (size_t i) { params_fp[i] = 0; });
    parallel_for_gpu(m_inference_stream[gpu], depth[gpu].n_elements(), [params_fp=depth[gpu].data()] __device__ (size_t i) { params_fp[i] = 0;});
    parallel_for_gpu(m_inference_stream[gpu], image[gpu].n_elements(), [params_fp=image[gpu].data()] __device__ (size_t i) { params_fp[i] = 0;});
    parallel_for_gpu(m_inference_stream[gpu], alive_counter[gpu].n_elements(), [params_fp=alive_counter[gpu].data()] __device__ (size_t i) { params_fp[i] = 0;});
    parallel_for_gpu(m_inference_stream[gpu], rays_alive[gpu].n_elements(), [params_fp=rays_alive[gpu].data()] __device__ (size_t i) {params_fp[i] = 0;});
    parallel_for_gpu(m_inference_stream[gpu], rays_t[gpu].n_elements(), [params_fp=rays_t[gpu].data()] __device__ (size_t i) { params_fp[i] = 0; });

      while(running[gpu]){
        if(step[gpu]>=m_max_infer_steps){
          running[gpu] = false;
          continue;
        }
        if (step[gpu] == 0) {
          // init rays at first step
          init_step0<<<div_round_up(num_alive[gpu], m_num_thread), m_num_thread, 0, m_inference_stream[gpu]>>>(
              rays_alive[gpu].view(), rays_t[gpu].view(), num_alive[gpu], nears[gpu].view());
        } else {
          // initialize alive_couter's value with 0
          int tmp_value = 0;
          hipMemcpyAsync(&alive_counter[gpu].view()(0, 0), &tmp_value, 1 * sizeof(int),
                    hipMemcpyHostToDevice, m_inference_stream[gpu]);

          // remove dead rays and reallocate alive rays, to accelerate next ray
          // marching
          int new_i = i[gpu] % 2;
          int old_i = (i[gpu] + 1) % 2;
          kernel_compact_rays<<<div_round_up(num_alive[gpu], m_num_thread), m_num_thread, 0, m_inference_stream[gpu]>>>(
              num_alive[gpu], rays_alive[gpu].view(), rays_t[gpu].view(), alive_counter[gpu].view(),
              new_i, old_i);
          hipMemcpyAsync(&num_alive[gpu], &alive_counter[gpu].view()(0, 0), 1 * sizeof(int),
                    hipMemcpyDeviceToHost, m_inference_stream[gpu]);
        }
        if (num_alive[gpu] <= 0) {
          running[gpu] = false;  // exit loop if no alive rays
          continue;
        }

        // decide compact_steps
        int num_step = max(min(N / num_alive[gpu], 8), 1);
        // round it to the multiply of 128
        int step_x_alive = div_round_up(num_alive[gpu] * num_step, 128) * 128;

        // march rays
        kernel_march_rays<<<div_round_up(num_alive[gpu], m_num_thread), m_num_thread, 0, m_inference_stream[gpu]>>>(
            num_alive[gpu], num_step, rays_alive[gpu].view(), rays_t[gpu].view(), rays_o[gpu].view(),
            rays_d[gpu].view(), m_bound, m_dt_gamma, m_dg_cascade, m_dg_h,
            m_density_grid[gpu].data(), m_mean_density, nears[gpu].view(), fars[gpu].view(),
            xyzs[gpu].view(), dirs[gpu].view(), deltas[gpu].view(), m_perturb, i[gpu]);

        tcnn::linear_kernel(linear_transformer<float>, 0, m_inference_stream[gpu], step_x_alive * 3,
          1.0/(2 * m_bound), 0.5, xyzs[gpu].data(), xyzs[gpu].data());
        tcnn::linear_kernel(linear_transformer<float>, 0, m_inference_stream[gpu], step_x_alive * 3,
          0.5, 0.5, dirs[gpu].data(), dirs[gpu].data());
        concat_network_in_and_out<<<div_round_up(step_x_alive, m_num_thread),
                                    m_num_thread, 0, m_inference_stream[gpu]>>>(
            xyzs[gpu].view(), dirs[gpu].view(),
            network_input[gpu].view(), step_x_alive, xyzs[gpu].rows(), dirs[gpu].rows());
        // forward through the network
        m_nerf_network[gpu]->inference_mixed_precision_impl(
            m_inference_stream[gpu], network_input[gpu], network_output[gpu]);

        // decompose network output
        decompose_network_in_and_out<<<div_round_up(step_x_alive, m_num_thread),
                                      m_num_thread, 0, m_inference_stream[gpu]>>>(
            sigmas[gpu].view(), rgbs[gpu].view(), network_output[gpu].view(), step_x_alive,
            sigmas[gpu].rows(), rgbs[gpu].cols());
        if(m_density_scale!=1) matrix_multiply_1x1n<<<div_round_up(step_x_alive, m_num_thread), m_num_thread, 0, m_inference_stream[gpu]>>>(
            m_density_scale, step_x_alive, sigmas[gpu].view());

        // composite rays
        kernel_composite_rays<<<div_round_up(num_alive[gpu], m_num_thread), m_num_thread, 0, m_inference_stream[gpu]>>>(
            num_alive[gpu], num_step, rays_alive[gpu].view(), rays_t[gpu].view(), sigmas[gpu].view(),
            rgbs[gpu].view(), deltas[gpu].view(), weight_sum[gpu].view(), depth[gpu].view(),
            image[gpu].view(), i[gpu]);
        step[gpu] += num_step;
        i[gpu] += 1;
      }
      // std::cout << "get image and depth" << std::endl;
      // get final image and depth
      get_image_and_depth<<<div_round_up(N, m_num_thread), m_num_thread, 0, m_inference_stream[gpu]>>>(
          image[gpu].view(), depth[gpu].view(), nears[gpu].view(), fars[gpu].view(), weight_sum[gpu].view(),
          m_bg_color, N);

      int offset = gpu*N;
      hipMemcpyAsync(deep_h+offset, &depth[gpu].view()(0, 0), N * sizeof(float), hipMemcpyDeviceToHost), m_inference_stream[gpu];
      hipMemcpyAsync(image_h+3*offset, &image[gpu].view()(0, 0), N * sizeof(float) * 3, hipMemcpyDeviceToHost, m_inference_stream[gpu]);
      hipStreamSynchronize(m_inference_stream[gpu]);
    // }
    // for(uint64_t gpu=0;gpu<NGPU;gpu++){
      // #pragma omp parallel for num_threads(8)
      for (int i = 0; i < N ; i++) {
        int in_i  = gpu*N+i;
        int out_i = NGPU*i+gpu;
        us_depth[out_i] = (unsigned char) (255.0 * deep_h[in_i]);
        us_image[out_i*3] = (unsigned char) (255.0 * image_h[in_i*3]); 
        us_image[out_i*3+1] = (unsigned char) (255.0 * image_h[in_i*3+1]); 
        us_image[out_i*3+2] = (unsigned char) (255.0 * image_h[in_i*3+2]); 
      }
    // }
        }));}
    for(auto &thread: threads) thread.join();
    hipDeviceSynchronize();

  Image img(resolution[0], resolution[1], us_image, us_depth);
  return img;
}

void NerfRender::generate_rays(struct Camera cam, Eigen::Matrix<float, 4, 4> pos, int threadid=-1) {

  int N = resolution[0] * resolution[1] / NGPU;  // number of pixels
  // std::cout << "N: " << N << std::endl;

  Eigen::Matrix<float, 4, 4> new_pose = nerf_matrix_to_ngp(pos, m_scale);

  int grid_size = ((N + m_num_thread) / m_num_thread);

  if(threadid==-1) for(uint64_t gpu=0;gpu<NGPU;gpu++){
    hipSetDevice(gpu);
    set_rays_o<<<grid_size, m_num_thread, 0, m_inference_stream[gpu]>>>(rays_o[gpu].view(), new_pose.block<3, 1>(0, 3), N);
    set_rays_d<<<grid_size, m_num_thread, 0, m_inference_stream[gpu]>>>(rays_d[gpu].view(), cam, new_pose.block<3, 3>(0, 0), resolution[0], N, gpu);
  }else{
    set_rays_o<<<grid_size, m_num_thread, 0, m_inference_stream[threadid]>>>(rays_o[threadid].view(), new_pose.block<3, 1>(0, 3), N);
    set_rays_d<<<grid_size, m_num_thread, 0, m_inference_stream[threadid]>>>(rays_d[threadid].view(), cam, new_pose.block<3, 3>(0, 0), resolution[0], N, threadid);
  }
}

void NerfRender::generate_density_grid() {
  const uint32_t H = m_dg_h;
	const uint32_t H2= H*H;
	const uint32_t H3= H*H*H;
	const float decay = 0.95;
	std::vector<float> tmpV(m_dg_cascade*H3, 1.0/64);
  hipSetDevice(0);

  m_density_grid[0].resize_and_copy_from_host(tmpV,tmpV.size());
	std::cout << "dg size: " << m_density_grid[0].size() << std::endl;
	tcnn::GPUMatrixDynamic<float> xyzs(3,H3);
	tcnn::GPUMatrixDynamic<float> cas_xyzs(3,H3);
	tcnn::GPUMatrixDynamic<precision_t> density_out(16,H3);
	tcnn::GPUMatrixDynamic<float> tmp_density(1,H3);

	int block_size = H;
  int grid_size = 3*H3/block_size;

	init_xyzs <<<grid_size, block_size, 0, m_inference_stream[0]>>> (xyzs.data(), 3*H3);

	for(int cas=0;cas<m_dg_cascade;cas++){
		float bound = 1<<cas < m_bound ? 1<<cas : m_bound;
		float half_grid_size = bound/H;

		dd_scale   <<<grid_size, block_size, 0, m_inference_stream[0]>>> (xyzs.data(), cas_xyzs.data(), 3*H3, bound-half_grid_size);
		add_random <<<grid_size, block_size, 0, m_inference_stream[0]>>> (cas_xyzs.data(), m_rng[0], 3*H3, half_grid_size);

		// m_nerf_network->density(nullptr,cas_xyzs,density_out);

		dd_scale  <<<H2, H, 0, m_inference_stream[0]>>> (density_out.slice_rows(0, 1).data(), tmp_density.data(), H3, 0.001691);
	}

	dg_update <<<H2, H, 0, m_inference_stream[0]>>> (m_density_grid[0].data(), tmp_density.data(), decay, H3);

#if NGPU > 1
  m_density_grid[0].copy_to_host(tmpV);
  for(uint64_t gpu=1;gpu<NGPU;gpu++){
    hipSetDevice(gpu);
    m_density_grid[gpu].resize_and_copy_from_host(tmpV,tmpV.size());
  }
#endif
}

void NerfRender::load_snapshot(const std::string& filepath_string){
  tlog::info() << "Reading snapshot";
	auto config = load_network_config(filepath_string);
	if(!config.contains("snapshot")){
		throw std::runtime_error{"File " + filepath_string + " does not contain a snapshot."};
	}

	const auto& snapshot = config["snapshot"];
	// here not to check snapshot version, tmp==1

  std::vector<float> tmp_aabb(snapshot["aabb"].size(), 0);
  for(int i=0;i<snapshot["aabb"].size();i++){
    tmp_aabb[i] = snapshot["aabb"].at(i);
  }
	m_bound = snapshot.value("bound", m_bound);
	m_scale = snapshot.value("scale", m_scale);
  m_dg_cascade = snapshot.value("cascade", m_dg_cascade);
  m_dg_h = snapshot.value("density_grid_size", m_dg_h);
  m_mean_density = snapshot.value("mean_density", m_mean_density);
  std::vector<float> tmp_density_grid(snapshot["density_grid"].size(), 0);
  for(int i=0;i<snapshot["density_grid"].size();i++){
    tmp_density_grid[i] = snapshot["density_grid"].at(i);
  }
  tlog::info() << tmp_density_grid[66 * m_dg_h * m_dg_h + 66 * m_dg_h + 66] 
    << "\t" << tmp_density_grid[66 * m_dg_h * m_dg_h + 66 * m_dg_h + 67]
    << "\t" << tmp_density_grid[66 * m_dg_h * m_dg_h + 66 * m_dg_h + 68]
    << "\tDG";
  for(uint64_t gpu=0;gpu<NGPU;gpu++){
    hipSetDevice(gpu);
    m_aabb[gpu].resize_and_copy_from_host(tmp_aabb);
    m_density_grid[gpu].resize_and_copy_from_host(tmp_density_grid);
  }
  float host_data[3] = {0};
  hipMemcpy(host_data, &m_density_grid[0].data()[66 * m_dg_h * m_dg_h + 66 * m_dg_h + 66], 3 * sizeof(float), hipMemcpyDeviceToHost);
  tlog::info() << "density grid : " << host_data[0] << "\t" << host_data[1] << "\t" << host_data[2];

	if (m_density_grid[0].size() != m_dg_h * m_dg_h * m_dg_h * m_dg_cascade) {
		throw std::runtime_error{"Incompatible number of grid cascades."};
	}

	m_network_config_path = filepath_string;
	m_network_config = config;
}

NGP_NAMESPACE_END
