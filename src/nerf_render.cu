#include "hip/hip_runtime.h"
#include <nerf-cuda/nerf_render.h>
#include <nerf-cuda/common.h>
#include <nerf-cuda/common_device.cuh>
#include <nerf-cuda/nerf_network.h>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/gpu_memory.h>
#include <tiny-cuda-nn/gpu_matrix.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>

#include <json/json.hpp>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#include <fstream>
#include <set>
#include <vector>

using namespace Eigen;
using namespace tcnn;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN

json merge_parent_network_config(const json &child, const fs::path &child_filename) {
	if (!child.contains("parent")) {
		return child;
	}
	fs::path parent_filename = child_filename.parent_path() / std::string(child["parent"]);
	tlog::info() << "Loading parent network config from: " << parent_filename.str();
	std::ifstream f{parent_filename.str()};
	json parent = json::parse(f, nullptr, true, true);
	parent = merge_parent_network_config(parent, parent_filename);
	parent.merge_patch(child);
	return parent;
}

NerfRender::NerfRender()
{
    m_network_config = {
		{"encoding", {
			{"otype", "HashGrid"},
			{"n_levels", 16},
			{"n_features_per_level", 2},
			{"log2_hashmap_size", 19},
			{"base_resolution", 16},
		}},
		{"network", {
			{"otype", "FullyFusedMLP"},
			{"n_neurons", 64},
			{"n_layers", 2},
			{"activation", "ReLU"},
			{"output_activation", "None"},
		}},
	};
    CUDA_CHECK_THROW(hipStreamCreate(&m_inference_stream));
}

NerfRender::~NerfRender()
{
}

json NerfRender::load_network_config(const fs::path& network_config_path) {
	if (!network_config_path.empty()) {
		m_network_config_path = network_config_path;
	}

	tlog::info() << "Loading network config from: " << network_config_path;

	if (network_config_path.empty() || !network_config_path.exists()) {
		throw std::runtime_error{std::string{"Network config \""} + network_config_path.str() + "\" does not exist."};
	}

	json result;
	if (equals_case_insensitive(network_config_path.extension(), "json")) {
		std::ifstream f{network_config_path.str()};
		result = json::parse(f, nullptr, true, true);
		result = merge_parent_network_config(result, network_config_path);
	} else if (equals_case_insensitive(network_config_path.extension(), "msgpack")) {
		std::ifstream f{network_config_path.str(), std::ios::in | std::ios::binary};
		result = json::from_msgpack(f);
		// we assume parent pointers are already resolved in snapshots.
	}

	return result;
}

void NerfRender::reload_network_from_file(const std::string& network_config_path) {
    if (!network_config_path.empty()) {
		m_network_config_path = network_config_path;
	}
    m_network_config = load_network_config(m_network_config_path);
    reset_network();
    
    // Haoran 
    // Need to do
    // 1. Load pretrained model
    // 2. Generate Density Grid, which will be used by ray sampler strategy!!!
    // now we just initialize the weight randomly.
    m_nerf_network -> initialize_xavier_uniform();
}

void NerfRender::reset_network()
{
    // reset the random seed
    m_rng = default_rng_t{m_seed};

    // Default config
	json config = m_network_config;
    json& encoding_config = config["encoding"];
    json& network_config = config["network"];
    json& dir_encoding_config = config["dir_encoding"];
	json& rgb_network_config = config["rgb_network"];
    uint32_t n_dir_dims = 3;
    uint32_t n_pos_dims = 3;
	uint32_t n_extra_dims = 0;   // Now, it's set to zero but it needs furture check! By Hangkun, 2022/06/30

    // Automatically determine certain parameters if we're dealing with the (hash)grid encoding
	if (to_lower(encoding_config.value("otype", "OneBlob")).find("grid") != std::string::npos) {
		encoding_config["n_pos_dims"] = n_pos_dims;   // 3 dimenison input

		const uint32_t n_features_per_level = encoding_config.value("n_features_per_level", 2u);
        uint32_t m_num_levels = 16u;

		if (encoding_config.contains("n_features") && encoding_config["n_features"] > 0) {
			m_num_levels = (uint32_t)encoding_config["n_features"] / n_features_per_level;
		} else {
			m_num_levels = encoding_config.value("n_levels", 16u);
		}

		// m_level_stats.resize(m_num_levels);
		// m_first_layer_column_stats.resize(m_num_levels);

		const uint32_t log2_hashmap_size = encoding_config.value("log2_hashmap_size", 15);

		uint32_t m_base_grid_resolution = encoding_config.value("base_resolution", 0);
		if (!m_base_grid_resolution) {
			m_base_grid_resolution = 1u << ((log2_hashmap_size) / n_pos_dims);
			encoding_config["base_resolution"] = m_base_grid_resolution;
		}

		float desired_resolution = 2048.0f; // Desired resolution of the finest hashgrid level over the unit cube

		// Automatically determine suitable per_level_scale
		float m_per_level_scale = encoding_config.value("per_level_scale", 0.0f);
		if (m_per_level_scale <= 0.0f && m_num_levels > 1) {
			m_per_level_scale = std::exp(std::log(desired_resolution * (float) m_bound / (float) m_base_grid_resolution) / (m_num_levels-1));
			encoding_config["per_level_scale"] = m_per_level_scale;
		}

		tlog::info()
			<< "GridEncoding: "
			<< " Nmin=" << m_base_grid_resolution
			<< " b=" << m_per_level_scale
			<< " F=" << n_features_per_level
			<< " T=2^" << log2_hashmap_size
			<< " L=" << m_num_levels
			;
	}
    
    // Generate the network
    m_nerf_network = std::make_shared<NerfNetwork<precision_t>>(
			n_pos_dims,
			n_dir_dims,
			n_extra_dims,
			n_pos_dims + 1, // The offset of 1 comes from the dt member variable of NerfCoordinate. HACKY
			encoding_config,
			dir_encoding_config,
			network_config,
			rgb_network_config
	);
	
}

void NerfRender::render_frame(struct Camera cam, Eigen::Matrix<float, 4, 4> pos, Eigen::Vector2i resolution)
{
    // cam : parameters of cam
    // pos : camera external parameters
    // resolution : [Width, Height]


    int N = resolution[0] * resolution[1];   // number of pixels
    tcnn::GPUMatrixDynamic<float> rays_o(N,3,tcnn::RM);      // initial points corresponding to pixels, in world coordination
    tcnn::GPUMatrixDynamic<float> rays_d(N,3,tcnn::RM);      // direction corresponding to pixels, in world coordination
    generate_rays(cam, pos, resolution, rays_o, rays_d);

    // Zilong
    // Need to do
    // 1. ray sample. refer the instant-ngp paper for details of the sample strategy. You can do it from the easiest equally spaced sampling strategy to the ray marching strategy.
    // 2. infer and volume rendering.
    // please refer to
    //     1. https://github.com/ashawkey/torch-ngp/blob/main/nerf/renderer.py line 318 ~ 380. espically the functions raymarching.compact_rays, raymarching.march_rays and raymarching.composite_rays. These function are places in https://github.com/ashawkey/torch-ngp/tree/main/raymarching/src.
    // 这里没有指定返回，后续可以再讨论返回图片是以什么格式。先渲染出来再说
	// below is an example of inference!
    tcnn::GPUMatrixDynamic<float> network_input(m_nerf_network -> input_width(), 4096);
    tcnn::GPUMatrixDynamic<precision_t> network_output(m_nerf_network -> padded_output_width(), 4096);
    tcnn::pcg32 rng = tcnn::pcg32((uint64_t) 32);
    network_input.initialize_xavier_uniform(rng);
    network_output.initialize_xavier_uniform(rng);
    m_nerf_network -> inference_mixed_precision(network_input, network_output);
}


__global__ void set_rays_d(MatrixView<float> rays_d, struct Camera cam, Eigen::Matrix<float, 3, 3> pose, int W, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    float i = (tid % W) + 0.5;
    float j = (tid / W) + 0.5;
    
    float zs = 1;
    float xs = (i - cam.cx) / cam.fl_x * zs;
    float ys = (j - cam.cy) / cam.fl_y * zs;
    Eigen::Vector3f directions(xs, ys, zs);
    directions = directions / directions.norm();
    Eigen::Vector3f ray_d = pose * directions;
    
    if (tid < N){
        rays_d(tid, 0) = ray_d[0];
        rays_d(tid, 1) = ray_d[1];
        rays_d(tid, 2) = ray_d[2];
    }
}

__global__ void set_rays_o(MatrixView<float> rays_o, Eigen::Vector3f ray_o, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // rays_o = rays_o[..., None, :].expand_as(rays_d) # [B, N, 3] @ function get_rays
    if (tid < N){
        rays_o(tid, 0) = ray_o[0];
        rays_o(tid, 1) = ray_o[1];
        rays_o(tid, 2) = ray_o[2];
    }
}

Eigen::Matrix<float, 4, 4> nerf_matrix_to_ngp(Eigen::Matrix<float, 4, 4> pose, float scale = 0.33, Eigen::Vector3f offset = Eigen::Vector3f(0, 0, 0)) {
    Eigen::Matrix<float, 4, 4> new_pose;
    new_pose << pose(1, 0), -pose(1, 1), -pose(1, 2), pose(1, 3) * scale + offset[0],
	              pose(2, 0), -pose(2, 1), -pose(2, 2), pose(2, 3) * scale + offset[1],
                pose(0, 0), -pose(0, 1), -pose(0, 2), pose(0, 3) * scale + offset[2],
	                       0,          0 ,           0,                              1;
    return new_pose;
}

void NerfRender::generate_rays(struct Camera cam, Eigen::Matrix<float, 4, 4> pos, Eigen::Vector2i resolution, tcnn::GPUMatrixDynamic<float>& rays_o, tcnn::GPUMatrixDynamic<float>& rays_d) {
    // Weixuan
    // Generate rays according to the input
    // please refer to 
    //     1. https://github.com/ashawkey/torch-ngp/blob/main/nerf/provider.py function nerf_matrix_to_ngp
    //     2. https://github.com/ashawkey/torch-ngp/blob/main/nerf/utils.py function get_rays
    // use cuda to speed up
    
    int N = resolution[0] * resolution[1];   // number of pixels
    std::cout << "N: " << N << std::endl;
    
    Eigen::Matrix<float, 4, 4> new_pose = nerf_matrix_to_ngp(pos);
    
    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);

    tcnn::MatrixView<float> rays_o_view = rays_o.view();
    set_rays_o<<<grid_size, block_size>>>(rays_o_view, new_pose.block<1, 3>(0, 3), N);
    
    tcnn::MatrixView<float> rays_d_view = rays_d.view();
    set_rays_d<<<grid_size, block_size>>>(rays_d_view, cam, new_pose.block<3, 3>(0, 0), resolution[0], N);
    
    float host_data[3] = {0, 0, 0};
    
    hipMemcpy(host_data, &rays_o_view(0,0), 3 * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "rays_o[0, :]: " << host_data[0] << ", " << host_data[1] << ", " << host_data[2] << std::endl;
    
    hipMemcpy(host_data, &rays_d_view(0,0), 3 * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "rays_d[0, :]: " << host_data[0] << ", " << host_data[1] << ", " << host_data[2] << std::endl;
}

void NerfRender::generate_density_grid()
{   
    // Jiang Wei
    // once the pretrained model is loaded! we can generate the density grid.
}

NGP_NAMESPACE_END
